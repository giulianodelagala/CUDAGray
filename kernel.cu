#include "hip/hip_runtime.h"

#include <iostream>
#include <opencv2/opencv.hpp>
//#include <opencv2/imgcodecs/imgcodecs.hpp>

#include "hip/hip_runtime.h"
#include ""

using namespace std;
using namespace cv;

#define CHANNELS 3

void ImpError(hipError_t err)
{
	cout << hipGetErrorString(err); // << " en " << __FILE__ << __LINE__;
	exit(EXIT_FAILURE);
}

void Imprimir(float* A, int n)
{
	for (int i = 0; i < n; ++i)
		if (i < n) cout << A[i] << " ";
	cout << "\n";
}

__global__
void colorToGrayKernel(unsigned char* Pout, unsigned char* Pin, int width, int height)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < width && row < height)
	{
		int grey_offset = row * width + col;
		int rgb_offset = grey_offset * CHANNELS;
		unsigned char r = Pin[rgb_offset]; //red
		unsigned char g = Pin[rgb_offset + 1]; //green
		unsigned char b = Pin[rgb_offset + 2]; //blue

		Pout[grey_offset] = 0.21f * r + 0.71f * g + 0.07f * b;
	}; 
}

void colorToGray(unsigned char* Pout, unsigned char* Pin, int width, int height, int n)
{
	int size = n * sizeof(char);
	int size_in = size * 3;
	unsigned char* d_Pin;
	unsigned char* d_Pout;

	hipError_t err = hipSuccess;

	err = hipMalloc(&d_Pin, size_in);
	err = hipMalloc(&d_Pout, size);

	err = hipMemcpy(d_Pin, Pin, size_in, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(width/ 32), ceil(height/ 32), 1);
	dim3 dimBlock(32, 32, 1);
	colorToGrayKernel <<<dimGrid, dimBlock>>> (d_Pout, d_Pin, width, height);

	err = hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
		ImpError(err);

	hipFree(d_Pin); hipFree(d_Pout);
}

int main()
{
	int height, width;
	int n; //height * width
	unsigned char* Pin;
	unsigned char* Pout;

	//FileStorage file("salida.txt", FileStorage::WRITE);

	Mat image = imread("lena.tif");
	height = image.rows; width = image.cols;
	cout << "h" << height << "\n";

	n = height * width;

	Pin = new unsigned char[n*3];
	Pout = new unsigned char[n];

	Pin = image.data;
	cout << (int)Pin[0] << (int)Pin[1] << " " << (int)Pin[1024];

	colorToGray(Pout, Pin, width, height, n);

	cout << "\n" << (int)Pout[1];

	Mat salida(height, width, CV_8U, Pout);

	//file << "salida" << salida;

	//cout << salida;
	//imshow("Display window", image);
	//imshow("Display window", salida);
	//waitKey(0);
	imwrite("lena_gray.png", salida);

	//delete Pin;
	//delete Pout;
	//cout << image;
	return 0;
}